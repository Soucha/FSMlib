#include "hip/hip_runtime.h"
/* Copyright (c) Michal Soucha, 2016
*
* This file is part of FSMlib
*
* FSMlib is free software: you can redistribute it and/or modify it under
* the terms of the GNU General Public License as published by the Free Software
* Foundation, either version 3 of the License, or (at your option) any later
* version.
*
* FSMlib is distributed in the hope that it will be useful, but WITHOUT ANY
* WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
* A PARTICULAR PURPOSE. See the GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License along with
* FSMlib. If not, see <http://www.gnu.org/licenses/>.
*/
#include "FSMsequence.h"

#include "hip/hip_runtime.h"
#include ""

extern unsigned int * prescan(int N, state_t * devArrIn);

namespace FSMsequence {

#define THREADS_PER_BLOCK 512
#define MAX_BLOCKS 1024

#define IS_ERROR(error) isError(error, __FILE__, __LINE__, dev)
#define CHECK_ERROR(error) if (isError(error, __FILE__, __LINE__, dev)) return false;
#define RETURN_ON_ERROR(error) if (isError(error, __FILE__, __LINE__, dev)) return sequence_vec_t();

	struct dev_ptrs_t {
		output_t * StateOutput = nullptr;
		output_t * TransitionOutput = nullptr;
		state_t * NextState = nullptr;
		state_t * Mapping = nullptr;
		state_t * NextDistIdx = nullptr;
		state_t * DistinguishedCount = nullptr;
		input_t * Distinguishing = nullptr;
		
		// Parallel SF
		seq_len_t * DistSeqLen = nullptr;

		// Parallel Queue
		state_t * PrevIdx = nullptr, * PrevIdxLen = nullptr;
		state_t * Unchecked = nullptr, * LinkPrev = nullptr;
		input_t * LinkIn = nullptr;
	};

	static void freeCuda(dev_ptrs_t& dev);
	static bool isError(hipError_t error, const char *file, int line, dev_ptrs_t& dev) {
		if (error != hipSuccess) {
			ERROR_MESSAGE("%s in %s, line %d", hipGetErrorString(error), file, line);
			freeCuda(dev);
			return true;
		}
		return false;
	}

#if SEQUENCES_PERFORMANCE_TEST
	static hipEvent_t start, stop;
	extern float gpuLoadTime, gpuProcessTime, gpuTotalTime;
#endif // SEQUENCES_PERFORMANCE_TEST

	static bool initCuda(const unique_ptr<DFSM>& fsm, bool useQueue, dev_ptrs_t& dev) {
		state_t N = fsm->getNumberOfStates();
		input_t P = fsm->getNumberOfInputs();
		state_t M = ((N - 1) * N) / 2;
		if (fsm->isOutputState()) {
			CHECK_ERROR(hipMalloc((void**)&(dev.StateOutput), N * sizeof(output_t)));
			output_t * outputs = new output_t[N];
			for (state_t state = 0; state < N; state++) {
				outputs[state] = fsm->getOutput(state, STOUT_INPUT);
			}
			CHECK_ERROR(hipMemcpy(dev.StateOutput, outputs, N*sizeof(output_t), hipMemcpyHostToDevice));
			delete outputs;
		}
		if (fsm->isOutputTransition()) {
			CHECK_ERROR(hipMalloc((void**)&(dev.TransitionOutput), N * P * sizeof(output_t)));
			output_t * outputs = new output_t[N*P];
			for (state_t state = 0; state < N; state++) {
				for (input_t input = 0; input < P; input++) {
					outputs[state*P + input] = fsm->getOutput(state, input);
				}
			}
			CHECK_ERROR(hipMemcpy(dev.TransitionOutput, outputs, N*P*sizeof(output_t), hipMemcpyHostToDevice));
			delete outputs;
		}
		CHECK_ERROR(hipMalloc((void**)&(dev.NextState), N * P * sizeof(state_t)));
		state_t * nextStates = new state_t[N*P];
		for (state_t state = 0; state < N; state++) {
			for (input_t input = 0; input < P; input++) {
				nextStates[state*P + input] = fsm->getNextState(state, input);
			}
		}
		CHECK_ERROR(hipMemcpy(dev.NextState, nextStates, N*P*sizeof(state_t), hipMemcpyHostToDevice));
		delete nextStates;

		CHECK_ERROR(hipMalloc((void**)&(dev.NextDistIdx), M*sizeof(state_t)));
		CHECK_ERROR(hipMemset(dev.NextDistIdx, int(NULL_STATE), M*sizeof(state_t)));
		CHECK_ERROR(hipMalloc((void**)&(dev.Distinguishing), M*sizeof(input_t)));

		CHECK_ERROR(hipMalloc((void**)&(dev.Mapping), M*sizeof(state_t)));
		state_t * mapping = new state_t[M];
		state_t idx = 0;
		for (state_t i = 0; i < N - 1; i++) {
			for (state_t j = i + 1; j < N; j++) {
				mapping[idx++] = i;
			}
		}
		CHECK_ERROR(hipMemcpy(dev.Mapping, mapping, M*sizeof(state_t), hipMemcpyHostToDevice));
		delete mapping;

		CHECK_ERROR(hipMalloc((void**)&(dev.DistinguishedCount), sizeof(state_t)));
		CHECK_ERROR(hipMemset(dev.DistinguishedCount, 0, sizeof(state_t)));
		
		if (useQueue) {// Queue
			CHECK_ERROR(hipMalloc((void**)&(dev.Unchecked), M*sizeof(state_t)));
			CHECK_ERROR(hipMalloc((void**)&(dev.PrevIdx), (M + 1)*sizeof(state_t)));
			CHECK_ERROR(hipMemset(dev.PrevIdx, 0, (M + 1)*sizeof(state_t)));
		} else {// SF
			CHECK_ERROR(hipMalloc((void**)&(dev.DistSeqLen), M*sizeof(seq_len_t)));
			CHECK_ERROR(hipMemset(dev.DistSeqLen, 0, M*sizeof(seq_len_t)));
		}
		return true;
	}

#define CUDA_FREE(ptr) if (ptr) {hipFree(ptr); ptr = nullptr;}

	static void freeCuda(dev_ptrs_t& dev) {
		CUDA_FREE(dev.NextState);
		CUDA_FREE(dev.StateOutput);
		CUDA_FREE(dev.TransitionOutput);
		CUDA_FREE(dev.NextDistIdx);
		CUDA_FREE(dev.Distinguishing);
		CUDA_FREE(dev.Mapping);
		// Queue
		CUDA_FREE(dev.PrevIdx);
		CUDA_FREE(dev.Unchecked);
		CUDA_FREE(dev.LinkIn);
		CUDA_FREE(dev.LinkPrev);
		CUDA_FREE(dev.PrevIdxLen);			
		// SF
		CUDA_FREE(dev.DistSeqLen);
	}

	// <--- SF's kernels --->

	__global__ void distinguishByStateOutputs(state_t M, state_t N, input_t P, state_t * distinguishedCount,
		state_t * mapping, state_t * nextDistIdx, input_t * distinguishing, state_t * distSeqLen, output_t * output) {
		state_t idx = threadIdx.x + blockIdx.x * blockDim.x;
		int distinguished = 0;
		if (idx < M) {
			state_t i = mapping[idx];
			state_t j = idx + 1 + i*(i + 3) / 2 - i*N;
			if (output[i] != output[j]) {
				distinguishing[idx] = STOUT_INPUT;
				nextDistIdx[idx] = idx;
				distSeqLen[idx] = 1;
				distinguished = 1;
			}
		}
		distinguished = __syncthreads_count(distinguished);
		if (threadIdx.x == 0) {
			atomicAdd(distinguishedCount, state_t(distinguished));
		}
	}

	__global__ void distinguishByTransitionOutputs(state_t M, state_t N, input_t P, state_t * distinguishedCount,
		state_t * mapping, state_t * nextDistIdx, input_t * distinguishing, state_t * distSeqLen, output_t * output) {
		state_t idx = threadIdx.x + blockIdx.x * blockDim.x;
		int distinguished = 0;
		if ((idx < M) && (nextDistIdx[idx] == NULL_STATE)) {
			state_t i = mapping[idx];
			state_t j = idx + 1 + i*(i + 3) / 2 - i*N;
			for (input_t input = 0; input < P; input++) {
				//printf("%d on %d: %d->%d %d->%d\n", idx, input, i, output[i*P + input], j, output[j*P + input]);
				if (output[i*P + input] != output[j*P + input]) {
					distinguishing[idx] = input;
					nextDistIdx[idx] = idx;
					distSeqLen[idx] = 1;
					distinguished = 1;
					break;
				}
			}
		}
		distinguished = __syncthreads_count(distinguished);
		if (threadIdx.x == 0) {
			atomicAdd(distinguishedCount, state_t(distinguished));
		}
	}

	__global__ void distinguishByNextStates(state_t M, state_t N, input_t P, seq_len_t len, state_t * distinguishedCount,
		state_t * mapping, state_t * nextDistIdx, input_t * distinguishing, state_t * distSeqLen, state_t * nextState) {
		state_t idx = threadIdx.x + blockIdx.x * blockDim.x;
		int distinguished = 0;
		if ((idx < M) && (nextDistIdx[idx] == NULL_STATE)) {
			state_t i = mapping[idx];
			state_t j = idx + 1 + i*(i + 3) / 2 - i*N;
			state_t nextStateI, nextStateJ, nextIdx;
			for (input_t input = 0; input < P; input++) {
				nextStateI = nextState[i*P + input];
				nextStateJ = nextState[j*P + input];
				//printf("%d on %d: %d->%d %d->%d\n", idx, input, i, nextStateI, j, nextStateJ);
				if (nextStateI != nextStateJ) {
					nextIdx = (nextStateI < nextStateJ) ?
						(nextStateI * N + nextStateJ - 1 - (nextStateI * (nextStateI + 3)) / 2) :
						(nextStateJ * N + nextStateI - 1 - (nextStateJ * (nextStateJ + 3)) / 2);
					if ((nextDistIdx[nextIdx] != NULL_STATE) && (distSeqLen[nextIdx] == len)) {
						distinguishing[idx] = input;
						nextDistIdx[idx] = nextIdx;
						distSeqLen[idx] = len + 1;
						distinguished = 1;
						break;
					}
				}
			}
		}
		distinguished = __syncthreads_count(distinguished);
		if (threadIdx.x == 0) {
			atomicAdd(distinguishedCount, state_t(distinguished));
		}
	}

	// <--- Queue's kernels --->

	__global__ void distinguishByOutputOrLink(state_t M, state_t N, input_t P, state_t * distinguishedCount,
		state_t * mapping, state_t * nextDistIdx, input_t * distinguishing,
		output_t * stateOutput, output_t * transitionOutput, state_t * nextState, state_t * unchecked, state_t * prevIdx) {

		state_t idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx < M) {
			state_t i = mapping[idx];
			state_t j = idx + 1 + i*(i + 3) / 2 - i*N;
			if ((stateOutput) && (stateOutput[i] != stateOutput[j])) {
				distinguishing[idx] = STOUT_INPUT;
				nextDistIdx[idx] = idx;
				int uncheckedIdx = atomicAdd(distinguishedCount, 1);
				unchecked[uncheckedIdx] = idx;
			}
			else {
				int distinguished = 0;
				if (transitionOutput) {
					for (input_t input = 0; input < P; input++) {
						//printf("%d on %d: %d->%d %d->%d\n", idx, input, i, output[i*P + input], j, output[j*P + input]);
						if (transitionOutput[i*P + input] != transitionOutput[j*P + input]) {
							distinguishing[idx] = input;
							nextDistIdx[idx] = idx;
							int uncheckedIdx = atomicAdd(distinguishedCount, 1);
							unchecked[uncheckedIdx] = idx;
							distinguished = 1;
							break;
						}
					}
				}
				if (!distinguished) {
					state_t nextStateI, nextStateJ, nextIdx;
					for (input_t input = 0; input < P; input++) {
						nextStateI = nextState[i*P + input];
						nextStateJ = nextState[j*P + input];
						//printf("%d on %d: %d->%d %d->%d\n", idx, input, i, nextStateI, j, nextStateJ);
						if (nextStateI != nextStateJ) {
							nextIdx = (nextStateI < nextStateJ) ?
								(nextStateI * N + nextStateJ - 1 - (nextStateI * (nextStateI + 3)) / 2) :
								(nextStateJ * N + nextStateI - 1 - (nextStateJ * (nextStateJ + 3)) / 2);
							if (idx != nextIdx) {
								atomicAdd(prevIdx + nextIdx, 1);
							}
						}
					}
				}
			}
		}
	}
	
	__global__ void prevStateLink(state_t M, state_t N, input_t P,
		state_t * mapping, state_t * nextDistIdx, state_t * nextState,
		state_t * prevIdx, state_t * prevIdxLen, input_t * linkIn, state_t * linkPrev) {

		state_t idx = threadIdx.x + blockIdx.x * blockDim.x;
		if ((idx < M) && (nextDistIdx[idx] == NULL_STATE)) {
			state_t i = mapping[idx];
			state_t j = idx + 1 + i*(i + 3) / 2 - i*N;
			state_t nextStateI, nextStateJ, nextIdx;
			for (input_t input = 0; input < P; input++) {
				nextStateI = nextState[i*P + input];
				nextStateJ = nextState[j*P + input];
				//printf("%d on %d: %d->%d %d->%d\n", idx, input, i, nextStateI, j, nextStateJ);
				if (nextStateI != nextStateJ) {
					nextIdx = (nextStateI < nextStateJ) ?
						(nextStateI * N + nextStateJ - 1 - (nextStateI * (nextStateI + 3)) / 2) :
						(nextStateJ * N + nextStateI - 1 - (nextStateJ * (nextStateJ + 3)) / 2);
					if (idx != nextIdx) {
						state_t basePrevIdx = atomicAdd(prevIdx + nextIdx, 1);
						basePrevIdx += prevIdxLen[nextIdx];
						linkIn[basePrevIdx] = input;
						linkPrev[basePrevIdx] = idx;
					}
				}
			}
		}
	}

	__global__ void processUnchecked(state_t M, state_t * distinguishedCount,
		state_t * nextDistIdx, input_t * distinguishing, state_t * unchecked,
		state_t * prevIdx, state_t * prevIdxLen, input_t * linkIn, state_t * linkPrev) {

		int base, count = 0;
		do {
			base = count;
			count = *distinguishedCount;
			__syncthreads();
			while (int(threadIdx.x) < count - base) {
				//printf("%d in %d (%d-%d)\n", threadIdx.x, count - base, count, base);
				state_t nextIdx = unchecked[base + threadIdx.x];
				state_t size = prevIdx[nextIdx];
				for (state_t k = 0; k < size; k++) {
					state_t prev = linkPrev[prevIdxLen[nextIdx] + k];
					state_t val = atomicCAS(nextDistIdx + prev, NULL_STATE, nextIdx);
					if (val == NULL_STATE) {
						distinguishing[prev] = linkIn[prevIdxLen[nextIdx] + k];
						int uncheckedIdx = atomicAdd(distinguishedCount, 1);
						unchecked[uncheckedIdx] = prev;
					}
				}
				base += blockDim.x;
			}
			__syncthreads();
		} while (*distinguishedCount < M);
	}

	// <--- common functions --->

	static sequence_vec_t getSequences(const unique_ptr<DFSM>& fsm, dev_ptrs_t& dev, state_t& M, bool useStout) {
#if SEQUENCES_PERFORMANCE_TEST
		RETURN_ON_ERROR(hipEventRecord(stop, 0));
		RETURN_ON_ERROR(hipEventSynchronize(stop));
		RETURN_ON_ERROR(hipEventElapsedTime(&gpuProcessTime, start, stop));
#endif // SEQUENCES_PERFORMANCE_TEST

		state_t *outNextDistIdx = new state_t[M];
		input_t * outDistinguishing = new input_t[M];
		hipMemcpy(outDistinguishing, dev.Distinguishing, M*sizeof(input_t), hipMemcpyDeviceToHost);
		hipMemcpy(outNextDistIdx, dev.NextDistIdx, M*sizeof(state_t), hipMemcpyDeviceToHost);

		state_t nextIdx;
		sequence_vec_t seq(M);
		for (state_t idx = 0; idx < M; idx++) {
			seq[idx].clear();
			nextIdx = idx;
			seq[idx].push_back(outDistinguishing[nextIdx]);
			while (nextIdx != outNextDistIdx[nextIdx]) {
				nextIdx = outNextDistIdx[nextIdx];
				if (outDistinguishing[nextIdx] != STOUT_INPUT) {
					if (useStout && seq[idx].back() != STOUT_INPUT) seq[idx].push_back(STOUT_INPUT);
					seq[idx].push_back(outDistinguishing[nextIdx]);
				} else if (fsm->isOutputTransition()) // filter last STOUT for Moore and DFA
					seq[idx].push_back(outDistinguishing[nextIdx]);
			}
		}
		delete outDistinguishing;
		delete outNextDistIdx;
		freeCuda(dev);

#if SEQUENCES_PERFORMANCE_TEST
		RETURN_ON_ERROR(hipEventRecord(stop, 0));
		RETURN_ON_ERROR(hipEventSynchronize(stop));
		RETURN_ON_ERROR(hipEventElapsedTime(&gpuTotalTime, start, stop));
		RETURN_ON_ERROR(hipEventDestroy(start));
		RETURN_ON_ERROR(hipEventDestroy(stop));
#endif // SEQUENCES_PERFORMANCE_TEST
		return seq;
	}

	sequence_vec_t getStatePairsShortestSeparatingSequences_ParallelSF(const unique_ptr<DFSM>& fsm, bool omitUnnecessaryStoutInputs) {
		RETURN_IF_UNREDUCED(fsm, "FSMsequence::getStatePairsShortestSeparatingSequences_ParallelSF", sequence_vec_t());
		state_t N = fsm->getNumberOfStates();
		input_t P = fsm->getNumberOfInputs();
		state_t M = ((N - 1) * N) / 2;
		if (M > MAX_BLOCKS * THREADS_PER_BLOCK) {
			ERROR_MESSAGE("%s::getStatePairsShortestSeparatingSequences_ParallelSF - too many states (%d), max is %d",
				machineTypeNames[fsm->getType()], M, MAX_BLOCKS * THREADS_PER_BLOCK);
			return sequence_vec_t();
		}
		dev_ptrs_t dev;
#if SEQUENCES_PERFORMANCE_TEST
		RETURN_ON_ERROR(hipEventCreate(&start));
		RETURN_ON_ERROR(hipEventCreate(&stop));
		RETURN_ON_ERROR(hipEventRecord(start, 0));
#endif // SEQUENCES_PERFORMANCE_TEST

		if (!initCuda(fsm, false, dev)) return sequence_vec_t();

#if SEQUENCES_PERFORMANCE_TEST
		RETURN_ON_ERROR(hipEventRecord(stop, 0));
		RETURN_ON_ERROR(hipEventSynchronize(stop));
		RETURN_ON_ERROR(hipEventElapsedTime(&gpuLoadTime, start, stop));
#endif // SEQUENCES_PERFORMANCE_TEST

		unsigned int threads, blocks;
		threads = (M < THREADS_PER_BLOCK) ? M : THREADS_PER_BLOCK;
		blocks = M / threads + (M % threads > 0);
		
#if SEQUENCES_PERFORMANCE_TEST
		RETURN_ON_ERROR(hipEventRecord(start, 0));
#endif // SEQUENCES_PERFORMANCE_TEST

		if (fsm->isOutputState()) {
			distinguishByStateOutputs<<<blocks, threads>>>(M, N, P, dev.DistinguishedCount,
				dev.Mapping, dev.NextDistIdx, dev.Distinguishing, dev.DistSeqLen, dev.StateOutput);
			RETURN_ON_ERROR(hipGetLastError());
			RETURN_ON_ERROR(hipDeviceSynchronize());
		}
		if (fsm->isOutputTransition()) {
			distinguishByTransitionOutputs<<<blocks, threads>>>(M, N, P, dev.DistinguishedCount,
				dev.Mapping, dev.NextDistIdx, dev.Distinguishing, dev.DistSeqLen, dev.TransitionOutput);
			RETURN_ON_ERROR(hipGetLastError());
			RETURN_ON_ERROR(hipDeviceSynchronize());
		}
		
		state_t count;
		hipMemcpy(&count, dev.DistinguishedCount, sizeof(state_t), hipMemcpyDeviceToHost);
		//printf("distinguished: %d\n", count);

		seq_len_t len = 0;
		while (count < M) {
			distinguishByNextStates<<<blocks, threads>>>(M, N, P, ++len, dev.DistinguishedCount,
				dev.Mapping, dev.NextDistIdx, dev.Distinguishing, dev.DistSeqLen, dev.NextState);
			RETURN_ON_ERROR(hipDeviceSynchronize());
			hipMemcpy(&count, dev.DistinguishedCount, sizeof(state_t), hipMemcpyDeviceToHost);
			//printf("distinguished: %d\n", count);
			//getchar();
		}
		return getSequences(fsm, dev, M, !omitUnnecessaryStoutInputs && fsm->isOutputState());
	}

	sequence_vec_t getStatePairsShortestSeparatingSequences_ParallelQueue(const unique_ptr<DFSM>& fsm, bool omitUnnecessaryStoutInputs) {
		RETURN_IF_UNREDUCED(fsm, "FSMsequence::getStatePairsShortestSeparatingSequences_ParallelQueue", sequence_vec_t());
		state_t N = fsm->getNumberOfStates();
		input_t P = fsm->getNumberOfInputs();
		state_t M = ((N - 1) * N) / 2;
		if (M > MAX_BLOCKS * THREADS_PER_BLOCK) {
			ERROR_MESSAGE("%s::getStatePairsShortestSeparatingSequences_ParallelQueue - too many states (%d), max is %d",
				machineTypeNames[fsm->getType()], M, MAX_BLOCKS * THREADS_PER_BLOCK);
			return sequence_vec_t();
		}
		dev_ptrs_t dev;
#if SEQUENCES_PERFORMANCE_TEST
		RETURN_ON_ERROR(hipEventCreate(&start));
		RETURN_ON_ERROR(hipEventCreate(&stop));
		RETURN_ON_ERROR(hipEventRecord(start, 0));
#endif // SEQUENCES_PERFORMANCE_TEST

		if (!initCuda(fsm, true, dev)) return sequence_vec_t();

#if SEQUENCES_PERFORMANCE_TEST
		RETURN_ON_ERROR(hipEventRecord(stop, 0));
		RETURN_ON_ERROR(hipEventSynchronize(stop));
		RETURN_ON_ERROR(hipEventElapsedTime(&gpuLoadTime, start, stop));
#endif // SEQUENCES_PERFORMANCE_TEST

		unsigned int threads, blocks;
		threads = (M < THREADS_PER_BLOCK) ? M : THREADS_PER_BLOCK;
		blocks = M / threads + (M % threads > 0);

#if SEQUENCES_PERFORMANCE_TEST
		RETURN_ON_ERROR(hipEventRecord(start, 0));
#endif // SEQUENCES_PERFORMANCE_TEST

		distinguishByOutputOrLink<<<blocks, threads>>>(M, N, P, dev.DistinguishedCount,
				dev.Mapping, dev.NextDistIdx, dev.Distinguishing,
				dev.StateOutput, dev.TransitionOutput, dev.NextState, dev.Unchecked, dev.PrevIdx);
		RETURN_ON_ERROR(hipGetLastError());
		RETURN_ON_ERROR(hipDeviceSynchronize());

		state_t count;
		hipMemcpy(&count, dev.DistinguishedCount, sizeof(state_t), hipMemcpyDeviceToHost);
		//printf("distinguished: %d\n", count);

		if (count < M) {
#if DEBUG
			state_t * tmp = (state_t*)malloc((M + 1)*sizeof(state_t));
			hipMemcpy(tmp, dev.PrevIdx, (M + 1)*sizeof(state_t), hipMemcpyDeviceToHost);
			for (int i = 0; i <= M; i++) {
				printf("%d ", tmp[i]);
			}printf("\n");
			getchar();
#endif
			dev.PrevIdxLen = prescan(M + 1, dev.PrevIdx);
			if (!dev.PrevIdxLen) {
				freeCuda(dev);
				return sequence_vec_t();
			}
#if DEBUG
			hipMemcpy(tmp, dev.PrevIdxLen, (M + 1)*sizeof(state_t), hipMemcpyDeviceToHost);
			for (int i = 0; i <= M; i++) {
				printf("%d ", tmp[i]);
			}printf("\n");
			getchar();
#endif
			state_t linkSize;
			hipMemcpy(&linkSize, dev.PrevIdxLen + M, sizeof(state_t), hipMemcpyDeviceToHost);

			RETURN_ON_ERROR(hipMemset(dev.PrevIdx, 0, M*sizeof(state_t)));
			RETURN_ON_ERROR(hipMalloc((void**)&(dev.LinkIn), linkSize*sizeof(input_t)));
			RETURN_ON_ERROR(hipMalloc((void**)&(dev.LinkPrev), linkSize*sizeof(state_t)));

			prevStateLink<<<blocks, threads>>>(M, N, P, dev.Mapping, dev.NextDistIdx,
				dev.NextState, dev.PrevIdx, dev.PrevIdxLen, dev.LinkIn, dev.LinkPrev);
			RETURN_ON_ERROR(hipGetLastError());
			RETURN_ON_ERROR(hipDeviceSynchronize());
#if DEBUG
			hipMemcpy(tmp, dev.PrevIdx, (M + 1)*sizeof(state_t), hipMemcpyDeviceToHost);
			for (int i = 0; i <= M; i++) {
				printf("%d ", tmp[i]);
			}printf("\n");
			hipMemcpy(tmp, dev.PrevIdxLen, (M + 1)*sizeof(state_t), hipMemcpyDeviceToHost);
			for (int i = 0; i <= M; i++) {
				printf("%d ", tmp[i]);
			}printf("\n");
			getchar();
			free(tmp);

			tmp = (state_t*)malloc(linkSize*sizeof(state_t));
			hipMemcpy(tmp, dev.LinkPrev, linkSize*sizeof(state_t), hipMemcpyDeviceToHost);
			for (int i = 0; i < linkSize; i++) {
				printf("%d ", tmp[i]);
			}printf("\n");
			hipMemcpy(tmp, dev.LinkIn, linkSize*sizeof(state_t), hipMemcpyDeviceToHost);
			for (int i = 0; i < linkSize; i++) {
				printf("%d ", tmp[i]);
			}printf("\n");
			getchar();
			free(tmp);
#endif
			processUnchecked<<<1, min(THREADS_PER_BLOCK, M)>>>(M, dev.DistinguishedCount,
				dev.NextDistIdx, dev.Distinguishing, dev.Unchecked, dev.PrevIdx, dev.PrevIdxLen, dev.LinkIn, dev.LinkPrev);
			RETURN_ON_ERROR(hipGetLastError());
#if DEBUG
			hipMemcpy(&count, dev.DistinguishedCount, sizeof(state_t), hipMemcpyDeviceToHost);
			printf("distinguished: %d\n", count);
			getchar();
#endif
		}
		return getSequences(fsm, dev, M, !omitUnnecessaryStoutInputs && fsm->isOutputState());
	}
}
