#include "hip/hip_runtime.h"
/* Copyright (c) Michal Soucha, 2016
*
* This file is part of FSMlib
*
* FSMlib is free software: you can redistribute it and/or modify it under
* the terms of the GNU General Public License as published by the Free Software
* Foundation, either version 3 of the License, or (at your option) any later
* version.
*
* FSMlib is distributed in the hope that it will be useful, but WITHOUT ANY
* WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
* A PARTICULAR PURPOSE. See the GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License along with
* FSMlib. If not, see <http://www.gnu.org/licenses/>.
*/
#include "FSMsequence.h"

#include "hip/hip_runtime.h"
#include ""

extern unsigned int * prescan(int N, state_t * devArrIn);

namespace FSMsequence {

#define THREADS_PER_BLOCK 512
#define MAX_BLOCKS 1024

#define IS_ERROR(error) isError(error, __FILE__, __LINE__)
#define CHECK_ERROR(error) if (isError(error, __FILE__, __LINE__)) return false;
#define RETURN_ON_ERROR(error) if (isError(error, __FILE__, __LINE__)) return;

	static void freeCuda();
	static bool isError(hipError_t error, const char *file, int line) {
		if (error != hipSuccess) {
			ERROR_MESSAGE("%s in %s, line %d", hipGetErrorString(error), file, line);
			freeCuda();
			return true;
		}
		return false;
	}

	static output_t * devStateOutput = NULL;
	static output_t * devTransitionOutput = NULL;
	static state_t * devNextState = NULL;
	static state_t * devMapping = NULL;
	static state_t * devNextDistIdx = NULL, *outNextDistIdx = NULL;
	static state_t * devDistinguishedCount = NULL;
	static input_t P, *devDistinguishing = NULL, *outDistinguishing = NULL;
	static state_t N, M;

	// Parallel SF
	static seq_len_t * devDistSeqLen = NULL;
	
	// Parallel Queue
	static state_t * devPrevIdx = NULL, *devPrevIdxLen = NULL;
	static state_t * devUnchecked = NULL, *devLinkPrev = NULL;
	static input_t *devLinkIn = NULL;

#if SEQUENCES_PERFORMANCE_TEST
	static hipEvent_t start, stop;
	extern float gpuLoadTime, gpuProcessTime, gpuTotalTime;
#endif // SEQUENCES_PERFORMANCE_TEST

	extern state_t getIdx(vector<state_t>& states, state_t stateId);

	static bool initCuda(DFSM* fsm, bool useQueue) {
		auto states = fsm->getStates();
		if (fsm->isOutputState()) {
			CHECK_ERROR(hipMalloc((void**)&(devStateOutput), N * sizeof(output_t)));
			output_t * outputs = new output_t[N];
			for (state_t state = 0; state < N; state++) {
				outputs[state] = fsm->getOutput(states[state], STOUT_INPUT);
			}
			CHECK_ERROR(hipMemcpy(devStateOutput, outputs, N*sizeof(output_t), hipMemcpyHostToDevice));
			delete outputs;
		}
		if (fsm->isOutputTransition()) {
			CHECK_ERROR(hipMalloc((void**)&(devTransitionOutput), N * P * sizeof(output_t)));
			output_t * outputs = new output_t[N*P];
			for (state_t state = 0; state < N; state++) {
				for (input_t input = 0; input < P; input++) {
					outputs[state*P + input] = fsm->getOutput(states[state], input);
				}
			}
			CHECK_ERROR(hipMemcpy(devTransitionOutput, outputs, N*P*sizeof(output_t), hipMemcpyHostToDevice));
			delete outputs;
		}
		CHECK_ERROR(hipMalloc((void**)&(devNextState), N * P * sizeof(state_t)));
		state_t * nextStates = new state_t[N*P];
		for (state_t state = 0; state < N; state++) {
			for (input_t input = 0; input < P; input++) {
				nextStates[state*P + input] = getIdx(states, fsm->getNextState(states[state], input));
			}
		}
		CHECK_ERROR(hipMemcpy(devNextState, nextStates, N*P*sizeof(state_t), hipMemcpyHostToDevice));
		delete nextStates;

		CHECK_ERROR(hipMalloc((void**)&(devNextDistIdx), M*sizeof(state_t)));
		CHECK_ERROR(hipMemset(devNextDistIdx, int(NULL_STATE), M*sizeof(state_t)));
		outNextDistIdx = new state_t[M];
		CHECK_ERROR(hipMalloc((void**)&(devDistinguishing), M*sizeof(input_t)));

		outDistinguishing = new input_t[M];

		CHECK_ERROR(hipMalloc((void**)&(devMapping), M*sizeof(state_t)));
		state_t * mapping = new state_t[M];
		state_t idx = 0;
		for (state_t i = 0; i < N - 1; i++) {
			for (state_t j = i + 1; j < N; j++) {
				mapping[idx++] = i;
			}
		}
		CHECK_ERROR(hipMemcpy(devMapping, mapping, M*sizeof(state_t), hipMemcpyHostToDevice));
		delete mapping;

		CHECK_ERROR(hipMalloc((void**)&(devDistinguishedCount), sizeof(state_t)));
		CHECK_ERROR(hipMemset(devDistinguishedCount, 0, sizeof(state_t)));
		
		if (useQueue) {// Queue
			CHECK_ERROR(hipMalloc((void**)&(devUnchecked), M*sizeof(state_t)));
			CHECK_ERROR(hipMalloc((void**)&(devPrevIdx), (M + 1)*sizeof(state_t)));
			CHECK_ERROR(hipMemset(devPrevIdx, 0, (M + 1)*sizeof(state_t)));
		} else {// SF
			CHECK_ERROR(hipMalloc((void**)&(devDistSeqLen), M*sizeof(seq_len_t)));
			CHECK_ERROR(hipMemset(devDistSeqLen, 0, M*sizeof(seq_len_t)));
		}
		return true;
	}

#define CUDA_FREE(ptr) if (ptr) {hipFree(ptr); ptr = NULL;}

	static void freeCuda() {
		CUDA_FREE(devNextState);
		CUDA_FREE(devStateOutput);
		CUDA_FREE(devTransitionOutput);
		CUDA_FREE(devNextDistIdx);
		CUDA_FREE(devDistinguishing);
		CUDA_FREE(devMapping);
		// Queue
		CUDA_FREE(devPrevIdx);
		CUDA_FREE(devUnchecked);
		CUDA_FREE(devLinkIn);
		CUDA_FREE(devLinkPrev);
		CUDA_FREE(devPrevIdxLen);			
		// SF
		CUDA_FREE(devDistSeqLen);
		
		if (outDistinguishing) {
			delete outDistinguishing;
			outDistinguishing = NULL;
		}
		if (outNextDistIdx) {
			delete outNextDistIdx;
			outNextDistIdx = NULL;
		}
	}

	// <--- SF's kernels --->

	__global__ void distinguishByStateOutputs(state_t M, state_t N, input_t P, state_t * distinguishedCount,
		state_t * mapping, state_t * nextDistIdx, input_t * distinguishing, state_t * distSeqLen, output_t * output) {
		state_t idx = threadIdx.x + blockIdx.x * blockDim.x;
		int distinguished = 0;
		if (idx < M) {
			state_t i = mapping[idx];
			state_t j = idx + 1 + i*(i + 3) / 2 - i*N;
			if (output[i] != output[j]) {
				distinguishing[idx] = STOUT_INPUT;
				nextDistIdx[idx] = idx;
				distSeqLen[idx] = 1;
				distinguished = 1;
			}
		}
		distinguished = __syncthreads_count(distinguished);
		if (threadIdx.x == 0) {
			atomicAdd(distinguishedCount, state_t(distinguished));
		}
	}

	__global__ void distinguishByTransitionOutputs(state_t M, state_t N, input_t P, state_t * distinguishedCount,
		state_t * mapping, state_t * nextDistIdx, input_t * distinguishing, state_t * distSeqLen, output_t * output) {
		state_t idx = threadIdx.x + blockIdx.x * blockDim.x;
		int distinguished = 0;
		if ((idx < M) && (nextDistIdx[idx] == NULL_STATE)) {
			state_t i = mapping[idx];
			state_t j = idx + 1 + i*(i + 3) / 2 - i*N;
			for (input_t input = 0; input < P; input++) {
				//printf("%d on %d: %d->%d %d->%d\n", idx, input, i, output[i*P + input], j, output[j*P + input]);
				if (output[i*P + input] != output[j*P + input]) {
					distinguishing[idx] = input;
					nextDistIdx[idx] = idx;
					distSeqLen[idx] = 1;
					distinguished = 1;
					break;
				}
			}
		}
		distinguished = __syncthreads_count(distinguished);
		if (threadIdx.x == 0) {
			atomicAdd(distinguishedCount, state_t(distinguished));
		}
	}

	__global__ void distinguishByNextStates(state_t M, state_t N, input_t P, seq_len_t len, state_t * distinguishedCount,
		state_t * mapping, state_t * nextDistIdx, input_t * distinguishing, state_t * distSeqLen, state_t * nextState) {
		state_t idx = threadIdx.x + blockIdx.x * blockDim.x;
		int distinguished = 0;
		if ((idx < M) && (nextDistIdx[idx] == NULL_STATE)) {
			state_t i = mapping[idx];
			state_t j = idx + 1 + i*(i + 3) / 2 - i*N;
			state_t nextStateI, nextStateJ, nextIdx;
			for (input_t input = 0; input < P; input++) {
				nextStateI = nextState[i*P + input];
				nextStateJ = nextState[j*P + input];
				//printf("%d on %d: %d->%d %d->%d\n", idx, input, i, nextStateI, j, nextStateJ);
				if (nextStateI != nextStateJ) {
					nextIdx = (nextStateI < nextStateJ) ?
						(nextStateI * N + nextStateJ - 1 - (nextStateI * (nextStateI + 3)) / 2) :
						(nextStateJ * N + nextStateI - 1 - (nextStateJ * (nextStateJ + 3)) / 2);
					if ((nextDistIdx[nextIdx] != NULL_STATE) && (distSeqLen[nextIdx] == len)) {
						distinguishing[idx] = input;
						nextDistIdx[idx] = nextIdx;
						distSeqLen[idx] = len + 1;
						distinguished = 1;
						break;
					}
				}
			}
		}
		distinguished = __syncthreads_count(distinguished);
		if (threadIdx.x == 0) {
			atomicAdd(distinguishedCount, state_t(distinguished));
		}
	}

	// <--- Queue's kernels --->

	__global__ void distinguishByOutputOrLink(state_t M, state_t N, input_t P, state_t * distinguishedCount,
		state_t * mapping, state_t * nextDistIdx, input_t * distinguishing,
		output_t * stateOutput, output_t * transitionOutput, state_t * nextState, state_t * unchecked, state_t * prevIdx) {

		state_t idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx < M) {
			state_t i = mapping[idx];
			state_t j = idx + 1 + i*(i + 3) / 2 - i*N;
			if ((stateOutput != NULL) && (stateOutput[i] != stateOutput[j])) {
				distinguishing[idx] = STOUT_INPUT;
				nextDistIdx[idx] = idx;
				int uncheckedIdx = atomicAdd(distinguishedCount, 1);
				unchecked[uncheckedIdx] = idx;
			}
			else {
				int distinguished = 0;
				if (transitionOutput != NULL) {
					for (input_t input = 0; input < P; input++) {
						//printf("%d on %d: %d->%d %d->%d\n", idx, input, i, output[i*P + input], j, output[j*P + input]);
						if (transitionOutput[i*P + input] != transitionOutput[j*P + input]) {
							distinguishing[idx] = input;
							nextDistIdx[idx] = idx;
							int uncheckedIdx = atomicAdd(distinguishedCount, 1);
							unchecked[uncheckedIdx] = idx;
							distinguished = 1;
							break;
						}
					}
				}
				if (!distinguished) {
					state_t nextStateI, nextStateJ, nextIdx;
					for (input_t input = 0; input < P; input++) {
						nextStateI = nextState[i*P + input];
						nextStateJ = nextState[j*P + input];
						//printf("%d on %d: %d->%d %d->%d\n", idx, input, i, nextStateI, j, nextStateJ);
						if (nextStateI != nextStateJ) {
							nextIdx = (nextStateI < nextStateJ) ?
								(nextStateI * N + nextStateJ - 1 - (nextStateI * (nextStateI + 3)) / 2) :
								(nextStateJ * N + nextStateI - 1 - (nextStateJ * (nextStateJ + 3)) / 2);
							if (idx != nextIdx) {
								atomicAdd(prevIdx + nextIdx, 1);
							}
						}
					}
				}
			}
		}
	}
	
	__global__ void prevStateLink(state_t M, state_t N, input_t P,
		state_t * mapping, state_t * nextDistIdx, state_t * nextState,
		state_t * prevIdx, state_t * prevIdxLen, input_t * linkIn, state_t * linkPrev) {

		state_t idx = threadIdx.x + blockIdx.x * blockDim.x;
		if ((idx < M) && (nextDistIdx[idx] == NULL_STATE)) {
			state_t i = mapping[idx];
			state_t j = idx + 1 + i*(i + 3) / 2 - i*N;
			state_t nextStateI, nextStateJ, nextIdx;
			for (input_t input = 0; input < P; input++) {
				nextStateI = nextState[i*P + input];
				nextStateJ = nextState[j*P + input];
				//printf("%d on %d: %d->%d %d->%d\n", idx, input, i, nextStateI, j, nextStateJ);
				if (nextStateI != nextStateJ) {
					nextIdx = (nextStateI < nextStateJ) ?
						(nextStateI * N + nextStateJ - 1 - (nextStateI * (nextStateI + 3)) / 2) :
						(nextStateJ * N + nextStateI - 1 - (nextStateJ * (nextStateJ + 3)) / 2);
					if (idx != nextIdx) {
						state_t basePrevIdx = atomicAdd(prevIdx + nextIdx, 1);
						basePrevIdx += prevIdxLen[nextIdx];
						linkIn[basePrevIdx] = input;
						linkPrev[basePrevIdx] = idx;
					}
				}
			}
		}
	}

	__global__ void processUnchecked(state_t M, state_t * distinguishedCount,
		state_t * nextDistIdx, input_t * distinguishing, state_t * unchecked,
		state_t * prevIdx, state_t * prevIdxLen, input_t * linkIn, state_t * linkPrev) {

		int base, count = 0;
		do {
			base = count;
			count = *distinguishedCount;
			__syncthreads();
			while (int(threadIdx.x) < count - base) {
				//printf("%d in %d (%d-%d)\n", threadIdx.x, count - base, count, base);
				state_t nextIdx = unchecked[base + threadIdx.x];
				state_t size = prevIdx[nextIdx];
				for (state_t k = 0; k < size; k++) {
					state_t prev = linkPrev[prevIdxLen[nextIdx] + k];
					state_t val = atomicCAS(nextDistIdx + prev, NULL_STATE, nextIdx);
					if (val == NULL_STATE) {
						distinguishing[prev] = linkIn[prevIdxLen[nextIdx] + k];
						int uncheckedIdx = atomicAdd(distinguishedCount, 1);
						unchecked[uncheckedIdx] = prev;
					}
				}
				base += blockDim.x;
			}
			__syncthreads();
		} while (*distinguishedCount < M);
	}

	// <--- common functions --->

	static bool getSequences(DFSM * fsm, vector<sequence_in_t> & seq) {
#if SEQUENCES_PERFORMANCE_TEST
		CHECK_ERROR(hipEventRecord(stop, 0));
		CHECK_ERROR(hipEventSynchronize(stop));
		CHECK_ERROR(hipEventElapsedTime(&gpuProcessTime, start, stop));
#endif // SEQUENCES_PERFORMANCE_TEST

		hipMemcpy(outDistinguishing, devDistinguishing, M*sizeof(input_t), hipMemcpyDeviceToHost);
		hipMemcpy(outNextDistIdx, devNextDistIdx, M*sizeof(state_t), hipMemcpyDeviceToHost);

		state_t nextIdx;
		seq.resize(M);
		for (state_t idx = 0; idx < M; idx++) {
			seq[idx].clear();
			nextIdx = idx;
			seq[idx].push_back(outDistinguishing[nextIdx]);
			while (nextIdx != outNextDistIdx[nextIdx]) {
				nextIdx = outNextDistIdx[nextIdx];
				if (fsm->isOutputTransition() || (outDistinguishing[nextIdx] != STOUT_INPUT)) // filter last STOUT for Moore and DFA
					seq[idx].push_back(outDistinguishing[nextIdx]);
			}
		}

		freeCuda();

#if SEQUENCES_PERFORMANCE_TEST
		CHECK_ERROR(hipEventRecord(stop, 0));
		CHECK_ERROR(hipEventSynchronize(stop));
		CHECK_ERROR(hipEventElapsedTime(&gpuTotalTime, start, stop));
		CHECK_ERROR(hipEventDestroy(start));
		CHECK_ERROR(hipEventDestroy(stop));
#endif // SEQUENCES_PERFORMANCE_TEST
		return true;
	}

	void getStatePairsShortestSeparatingSequences_ParallelSF(DFSM * fsm, vector<sequence_in_t> & seq) {
		N = fsm->getNumberOfStates();
		P = fsm->getNumberOfInputs();
		M = ((N - 1) * N) / 2;
		seq.clear();
		if (M > MAX_BLOCKS * THREADS_PER_BLOCK) {
			ERROR_MESSAGE("%s::getStatePairsShortestSeparatingSequences_ParallelSF - too many states (%d), max is %d",
				machineTypeNames[fsm->getType()], M, MAX_BLOCKS * THREADS_PER_BLOCK);
			return; 
		}
#if SEQUENCES_PERFORMANCE_TEST
		RETURN_ON_ERROR(hipEventCreate(&start));
		RETURN_ON_ERROR(hipEventCreate(&stop));
		RETURN_ON_ERROR(hipEventRecord(start, 0));
#endif // SEQUENCES_PERFORMANCE_TEST

		if (!initCuda(fsm, false)) return;

#if SEQUENCES_PERFORMANCE_TEST
		RETURN_ON_ERROR(hipEventRecord(stop, 0));
		RETURN_ON_ERROR(hipEventSynchronize(stop));
		RETURN_ON_ERROR(hipEventElapsedTime(&gpuLoadTime, start, stop));
#endif // SEQUENCES_PERFORMANCE_TEST

		unsigned int threads, blocks;
		threads = (M < THREADS_PER_BLOCK) ? M : THREADS_PER_BLOCK;
		blocks = M / threads + (M % threads > 0);
		
#if SEQUENCES_PERFORMANCE_TEST
		RETURN_ON_ERROR(hipEventRecord(start, 0));
#endif // SEQUENCES_PERFORMANCE_TEST

		if (fsm->isOutputState()) {
			distinguishByStateOutputs<<<blocks, threads>>>(M, N, P, devDistinguishedCount,
				devMapping, devNextDistIdx, devDistinguishing, devDistSeqLen, devStateOutput);
			RETURN_ON_ERROR(hipGetLastError());
			RETURN_ON_ERROR(hipDeviceSynchronize());
		}
		if (fsm->isOutputTransition()) {
			distinguishByTransitionOutputs<<<blocks, threads>>>(M, N, P, devDistinguishedCount,
				devMapping, devNextDistIdx, devDistinguishing, devDistSeqLen, devTransitionOutput);
			RETURN_ON_ERROR(hipGetLastError());
			RETURN_ON_ERROR(hipDeviceSynchronize());
		}
		
		state_t count;
		hipMemcpy(&count, devDistinguishedCount, sizeof(state_t), hipMemcpyDeviceToHost);
		//printf("distinguished: %d\n", count);

		seq_len_t len = 0;
		while (count < M) {
			distinguishByNextStates<<<blocks, threads>>>(M, N, P, ++len, devDistinguishedCount,
				devMapping, devNextDistIdx, devDistinguishing, devDistSeqLen, devNextState);
			hipDeviceSynchronize();
			hipMemcpy(&count, devDistinguishedCount, sizeof(state_t), hipMemcpyDeviceToHost);
			//printf("distinguished: %d\n", count);
			//getchar();
		}

		if (!getSequences(fsm, seq)) seq.clear();
	}

	void getStatePairsShortestSeparatingSequences_ParallelQueue(DFSM * fsm, vector<sequence_in_t> & seq) {
		N = fsm->getNumberOfStates();
		P = fsm->getNumberOfInputs();
		M = ((N - 1) * N) / 2;
		seq.clear();
		if (M > MAX_BLOCKS * THREADS_PER_BLOCK) {
			ERROR_MESSAGE("%s::getStatePairsShortestSeparatingSequences_ParallelQueue - too many states (%d), max is %d",
				machineTypeNames[fsm->getType()], M, MAX_BLOCKS * THREADS_PER_BLOCK);
			return;
		}
#if SEQUENCES_PERFORMANCE_TEST
		RETURN_ON_ERROR(hipEventCreate(&start));
		RETURN_ON_ERROR(hipEventCreate(&stop));
		RETURN_ON_ERROR(hipEventRecord(start, 0));
#endif // SEQUENCES_PERFORMANCE_TEST

		if (!initCuda(fsm, true)) return;

#if SEQUENCES_PERFORMANCE_TEST
		RETURN_ON_ERROR(hipEventRecord(stop, 0));
		RETURN_ON_ERROR(hipEventSynchronize(stop));
		RETURN_ON_ERROR(hipEventElapsedTime(&gpuLoadTime, start, stop));
#endif // SEQUENCES_PERFORMANCE_TEST

		unsigned int threads, blocks;
		threads = (M < THREADS_PER_BLOCK) ? M : THREADS_PER_BLOCK;
		blocks = M / threads + (M % threads > 0);

#if SEQUENCES_PERFORMANCE_TEST
		RETURN_ON_ERROR(hipEventRecord(start, 0));
#endif // SEQUENCES_PERFORMANCE_TEST

		distinguishByOutputOrLink<<<blocks, threads>>>(M, N, P, devDistinguishedCount,
				devMapping, devNextDistIdx, devDistinguishing,
				devStateOutput, devTransitionOutput, devNextState, devUnchecked, devPrevIdx);
		RETURN_ON_ERROR(hipGetLastError());
		RETURN_ON_ERROR(hipDeviceSynchronize());

		state_t count;
		hipMemcpy(&count, devDistinguishedCount, sizeof(state_t), hipMemcpyDeviceToHost);
		//printf("distinguished: %d\n", count);

		if (count < M) {
#if DEBUG
			state_t * tmp = (state_t*)malloc((M + 1)*sizeof(state_t));
			hipMemcpy(tmp, devPrevIdx, (M + 1)*sizeof(state_t), hipMemcpyDeviceToHost);
			for (int i = 0; i <= M; i++) {
				printf("%d ", tmp[i]);
			}printf("\n");
			getchar();
#endif
			devPrevIdxLen = prescan(M + 1, devPrevIdx);
			if (devPrevIdxLen == NULL) {
				freeCuda();
				return;
			}
#if DEBUG
			hipMemcpy(tmp, devPrevIdxLen, (M + 1)*sizeof(state_t), hipMemcpyDeviceToHost);
			for (int i = 0; i <= M; i++) {
				printf("%d ", tmp[i]);
			}printf("\n");
			getchar();
#endif
			state_t linkSize;
			hipMemcpy(&linkSize, devPrevIdxLen + M, sizeof(state_t), hipMemcpyDeviceToHost);

			RETURN_ON_ERROR(hipMemset(devPrevIdx, 0, M*sizeof(state_t)));
			RETURN_ON_ERROR(hipMalloc((void**)&(devLinkIn), linkSize*sizeof(input_t)));
			RETURN_ON_ERROR(hipMalloc((void**)&(devLinkPrev), linkSize*sizeof(state_t)));

			prevStateLink<<<blocks, threads>>>(M, N, P, devMapping, devNextDistIdx,
				devNextState, devPrevIdx, devPrevIdxLen, devLinkIn, devLinkPrev);
			RETURN_ON_ERROR(hipGetLastError());
			RETURN_ON_ERROR(hipDeviceSynchronize());
#if DEBUG
			hipMemcpy(tmp, devPrevIdx, (M + 1)*sizeof(state_t), hipMemcpyDeviceToHost);
			for (int i = 0; i <= M; i++) {
				printf("%d ", tmp[i]);
			}printf("\n");
			hipMemcpy(tmp, devPrevIdxLen, (M + 1)*sizeof(state_t), hipMemcpyDeviceToHost);
			for (int i = 0; i <= M; i++) {
				printf("%d ", tmp[i]);
			}printf("\n");
			getchar();
			free(tmp);

			tmp = (state_t*)malloc(linkSize*sizeof(state_t));
			hipMemcpy(tmp, devLinkPrev, linkSize*sizeof(state_t), hipMemcpyDeviceToHost);
			for (int i = 0; i < linkSize; i++) {
				printf("%d ", tmp[i]);
			}printf("\n");
			hipMemcpy(tmp, devLinkIn, linkSize*sizeof(state_t), hipMemcpyDeviceToHost);
			for (int i = 0; i < linkSize; i++) {
				printf("%d ", tmp[i]);
			}printf("\n");
			getchar();
			free(tmp);
#endif
			processUnchecked<<<1, min(THREADS_PER_BLOCK, M)>>>(M, devDistinguishedCount,
				devNextDistIdx, devDistinguishing, devUnchecked, devPrevIdx, devPrevIdxLen, devLinkIn, devLinkPrev);
			RETURN_ON_ERROR(hipGetLastError());
#if DEBUG
			hipMemcpy(&count, devDistinguishedCount, sizeof(state_t), hipMemcpyDeviceToHost);
			printf("distinguished: %d\n", count);
			getchar();
#endif
		}

		if (!getSequences(fsm, seq)) seq.clear();
	}
}
