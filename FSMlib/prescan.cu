#include "hip/hip_runtime.h"
/* Copyright (c) Michal Soucha, 2016
*
* This file is part of FSMlib
*
* FSMlib is free software: you can redistribute it and/or modify it under
* the terms of the GNU General Public License as published by the Free Software
* Foundation, either version 3 of the License, or (at your option) any later
* version.
*
* FSMlib is distributed in the hope that it will be useful, but WITHOUT ANY
* WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
* A PARTICULAR PURPOSE. See the GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License along with
* FSMlib. If not, see <http://www.gnu.org/licenses/>.
*
*
*	Parallel sum
*	------------
*	The algorithm handles arbitrarily long data array.
*	It uses prescan function from http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html
*	with arrays devIdxs and devSums for storing indexes and meta-sums of indexes respectively.
*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string>

#include "hip/hip_runtime.h"
#include ""

#include "FSMlib.h"

#define THREADS_PER_BLOCK 1024 // need to be power of 2

#define NUM_BANKS 16  
#define LOG_NUM_BANKS 4  
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))


//#define DEBUG 0

const int blockElements = 2 * THREADS_PER_BLOCK;

static int blocks, levels, count, level, startIdx;
static unsigned int *blockCounts = NULL, *devIdxs = NULL, *devSums = NULL, devN;
#if DEBUG
static unsigned int  *idxs = NULL, *sums = NULL;
#endif
#define IS_ERROR(error) isError(error, __FILE__, __LINE__)
#define CHECK_ERROR(error) if (isError(error, __FILE__, __LINE__)) return false;

static void freeCuda(bool onError = true);
static bool isError(hipError_t error, const char *file, int line) {
	if (error != hipSuccess) {
		ERROR_MESSAGE("prescan: %s in %s at line %d\n", hipGetErrorString(error), file, line);
		freeCuda();
		return true;
	}
	return false;
}

int getLengthAsPowersTwoSum(int n) {
	return (n / blockElements)*blockElements + (1 << (int)ceilf(logf(n % blockElements) / logf(2)));
}

__global__ void prescan(unsigned int *g_odata, unsigned int *g_idata, unsigned int *g_udata, int n) {
	// modified function from http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html
	extern __shared__ int temp[];  // allocated on invocation  

	// set proper length of array in current block
	n = (n < (2 * blockDim.x * (blockIdx.x + 1))) ? (n - 2 * blockDim.x*blockIdx.x) : 2 * blockDim.x;

	int thid = threadIdx.x;
	if ((thid <= n / 2) && (n <= 1)) {
		g_udata[blockIdx.x] = g_idata[thid + 2 * blockDim.x*blockIdx.x];
		g_odata[thid + 2 * blockDim.x*blockIdx.x] = 0;
	}
	else if (thid < n / 2) {
		int offset = 1;
		int cai = thid;
		int cbi = thid + (n / 2);
		int bankOffsetA = CONFLICT_FREE_OFFSET(cai);
		int bankOffsetB = CONFLICT_FREE_OFFSET(cbi);
		temp[cai + bankOffsetA] = g_idata[cai + 2 * blockDim.x*blockIdx.x];// modification: data from the correct block
		temp[cbi + bankOffsetB] = g_idata[cbi + 2 * blockDim.x*blockIdx.x];
		for (int d = n >> 1; d > 0; d >>= 1) { // build sum in place up the tree  
			__syncthreads();
			if (thid < d) {
				int ai = offset*(2 * thid + 1) - 1;
				int bi = offset*(2 * thid + 2) - 1;
				ai += CONFLICT_FREE_OFFSET(ai);
				bi += CONFLICT_FREE_OFFSET(bi);
				temp[bi] += temp[ai];
			}
			offset *= 2;
		}
		if (thid == 0) {
			// store sum of block values
			g_udata[blockIdx.x] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
			temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
		}

		for (int d = 1; d < n; d *= 2) { // traverse down tree & build scan  
			offset >>= 1;
			__syncthreads();
			if (thid < d) {
				int ai = offset*(2 * thid + 1) - 1;
				int bi = offset*(2 * thid + 2) - 1;
				ai += CONFLICT_FREE_OFFSET(ai);
				bi += CONFLICT_FREE_OFFSET(bi);
				int t = temp[ai];
				temp[ai] = temp[bi];
				temp[bi] += t;
			}
		}
		__syncthreads();

		g_odata[cai + 2 * blockDim.x*blockIdx.x] = temp[cai + bankOffsetA];// modification: data to the correct block
		g_odata[cbi + 2 * blockDim.x*blockIdx.x] = temp[cbi + bankOffsetB];
	}
}

__global__ void add(unsigned int *out, unsigned int *in, int n) {
	int thid = threadIdx.x + blockDim.x* blockIdx.x;
	if (thid < n)
		out[thid] += in[blockIdx.x / 2];
}

static bool initCUDA() {
	CHECK_ERROR(hipMalloc((void**)&(devIdxs), devN*sizeof(unsigned int)));
	CHECK_ERROR(hipMemset(devIdxs, 0, devN*sizeof(unsigned int)));

	blockCounts = (unsigned int*)malloc(levels*sizeof(unsigned int));

	count = getLengthAsPowersTwoSum(blocks);
	blockCounts[0] = count;
	if ((count - 1) / blockElements >= 1) {// how many times must be prescan run
		int dx = count;
		int i = 1;
		while (dx > 1) {
			dx = dx / blockElements + (dx%blockElements>0);
			dx = getLengthAsPowersTwoSum(dx);
			blockCounts[i++] = dx;
			count += dx;
		}
	}
	else {
		count++;
		blockCounts[1] = 1;
	}
#if DEBUG
	for (int i = 0; i < levels; i++) {
		printf("%d ", blockCounts[i]);
	}printf("\n");
	getchar();

	idxs = (unsigned int*)malloc(devN*sizeof(unsigned int));
	sums = (unsigned int*)malloc(count*sizeof(unsigned int));
#endif
	CHECK_ERROR(hipMalloc((void**)&(devSums), count*sizeof(unsigned int)));
	CHECK_ERROR(hipMemset(devSums, 0, count*sizeof(unsigned int)));
	return true;
}

static bool callPrescan() {
	prescan<<<blocks, THREADS_PER_BLOCK, 2 * THREADS_PER_BLOCK*sizeof(unsigned int)>>>(devIdxs, devIdxs, devSums, devN);

#if DEBUG
	CHECK_ERROR(hipDeviceSynchronize());
	hipMemcpy(idxs, devIdxs, devN*sizeof(unsigned int), hipMemcpyDeviceToHost);
	for (int i = 0; i < devN; i++)	{
		printf("%d ", idxs[i]);
	}printf("\n");
	getchar();
#endif

	int dx, size, actBlocks;
	level = 0;
	count = blockCounts[level];
	startIdx = 0;
	for (level = 1; level < levels; level++) {// prescan of sums at each level
		dx = blockCounts[level];
		size = count - startIdx;
		actBlocks = size / blockElements + (size % blockElements > 0);
#if DEBUG
		printf("%d %d %d %d %d\n", dx, count, startIdx, size, actBlocks);
#endif
		CHECK_ERROR(hipDeviceSynchronize());
		prescan<<<actBlocks, THREADS_PER_BLOCK, 2 * THREADS_PER_BLOCK*sizeof(unsigned int)>>>(
			devSums + startIdx, devSums + startIdx, devSums + count, size);
		CHECK_ERROR(hipGetLastError());
		startIdx = count;
		count += dx;
	}

#if DEBUG
	hipMemcpy(sums, devSums, count*sizeof(unsigned int), hipMemcpyDeviceToHost);
	for (int i = 0; i < count; i++)
	{
		printf("%d ", sums[i]);
	}printf("\n");
	getchar();
#endif
	return true;
}

static bool callAdd(int N) {
	int dx;
	if (level > 1) {
		count -= blockCounts[--level];
		startIdx -= blockCounts[level - 1];
	}
	while (level > 1) {
		dx = blockCounts[--level];
		count -= dx;
		startIdx -= blockCounts[level - 1];
#if DEBUG
		printf("%d %d %d\n", dx, count, startIdx);
#endif
		CHECK_ERROR(hipDeviceSynchronize());
		add<<<2 * dx, THREADS_PER_BLOCK>>>(devSums + startIdx, devSums + count, count - startIdx);
		CHECK_ERROR(hipGetLastError());
	}
	if (blocks > 1) {// if all data are in one block, indexes are correct. otherwise add prescan sums
		CHECK_ERROR(hipDeviceSynchronize());
		add<<<2 * blocks, THREADS_PER_BLOCK>>>(devIdxs, devSums, N);
		CHECK_ERROR(hipGetLastError());
	}

#if DEBUG
	hipMemcpy(idxs, devIdxs, devN*sizeof(unsigned int), hipMemcpyDeviceToHost);
	for (int i = 0; i < devN; i++) {
		printf("%d ", idxs[i]);
	}printf("\n");
	getchar();
	hipMemcpy(sums, devSums, count*sizeof(unsigned int), hipMemcpyDeviceToHost);
	for (int i = 0; i < count; i++) {
		printf("%d ", sums[i]);
	}printf("\n");
	getchar();
#endif
	return true;
}

static void freeCuda(bool onError) {

#define CUDA_FREE(ptr) if (ptr) {hipFree(ptr); ptr = NULL;}
#define MEM_FREE(ptr) if (ptr) {free(ptr); ptr = NULL;}

	if (onError) CUDA_FREE(devIdxs);// cannot be freed because it is returned
	CUDA_FREE(devSums);

	if (levels > 1) MEM_FREE(blockCounts);

#if DEBUG
	MEM_FREE(sums);
	MEM_FREE(idxs);
#endif
}

unsigned int * prescan(int N, unsigned int * devArrIn) {
	blocks = (N % blockElements == 0) ? N / blockElements : N / blockElements + 1;
	levels = ceilf(logf(N) / logf(blockElements));
	devN = getLengthAsPowersTwoSum(N);

	if (!initCUDA()) return NULL;
	hipMemcpy(devIdxs, devArrIn, N*sizeof(unsigned int), hipMemcpyDeviceToDevice);

	if (!callPrescan()) return NULL;
	if (!callAdd(N)) return NULL;
	if (IS_ERROR(hipDeviceSynchronize())) return NULL;
	freeCuda(false);

	return devIdxs;
}

