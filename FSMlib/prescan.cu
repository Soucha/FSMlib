#include "hip/hip_runtime.h"
/* Copyright (c) Michal Soucha, 2016
*
* This file is part of FSMlib
*
* FSMlib is free software: you can redistribute it and/or modify it under
* the terms of the GNU General Public License as published by the Free Software
* Foundation, either version 3 of the License, or (at your option) any later
* version.
*
* FSMlib is distributed in the hope that it will be useful, but WITHOUT ANY
* WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
* A PARTICULAR PURPOSE. See the GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License along with
* FSMlib. If not, see <http://www.gnu.org/licenses/>.
*
*
*	Parallel sum
*	------------
*	The algorithm handles arbitrarily long data array.
*	It uses prescan function from http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html
*	with arrays devIdxs and devSums for storing indexes and meta-sums of indexes respectively.
*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string>

#include "hip/hip_runtime.h"
#include ""

#include "FSMlib.h"

#define THREADS_PER_BLOCK 1024 // need to be power of 2

#define NUM_BANKS 16  
#define LOG_NUM_BANKS 4  
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

//#define DEBUG 0

const int blockElements = 2 * THREADS_PER_BLOCK;

struct dev_prescan_t {
	int blocks, levels, count, startIdx;
	unsigned int *blockCounts = nullptr, *Idxs = nullptr, *Sums = nullptr, N;
};

#if DEBUG
static unsigned int  *idxs = nullptr, *sums = nullptr;
#endif
#define IS_ERROR(error) isError(error, __FILE__, __LINE__, dev)
#define CHECK_ERROR(error) if (isError(error, __FILE__, __LINE__, dev)) return false;

static void freeCuda(dev_prescan_t& dev, bool onError = true);
static bool isError(hipError_t error, const char *file, int line, dev_prescan_t& dev) {
	if (error != hipSuccess) {
		ERROR_MESSAGE("prescan: %s in %s at line %d\n", hipGetErrorString(error), file, line);
		freeCuda(dev);
		return true;
	}
	return false;
}

int getLengthAsPowersTwoSum(int n) {
	return (n / blockElements)*blockElements + (1 << (int)ceil(log((double)(n % blockElements)) / log(2)));
}

__global__ void prescan(unsigned int *g_odata, unsigned int *g_idata, unsigned int *g_udata, int n) {
	// modified function from http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html
	extern __shared__ int temp[];  // allocated on invocation  

	// set proper length of array in current block
	n = (n < (2 * blockDim.x * (blockIdx.x + 1))) ? (n - 2 * blockDim.x*blockIdx.x) : 2 * blockDim.x;

	int thid = threadIdx.x;
	if ((thid <= n / 2) && (n <= 1)) {
		g_udata[blockIdx.x] = g_idata[thid + 2 * blockDim.x*blockIdx.x];
		g_odata[thid + 2 * blockDim.x*blockIdx.x] = 0;
	}
	else if (thid < n / 2) {
		int offset = 1;
		int cai = thid;
		int cbi = thid + (n / 2);
		int bankOffsetA = CONFLICT_FREE_OFFSET(cai);
		int bankOffsetB = CONFLICT_FREE_OFFSET(cbi);
		temp[cai + bankOffsetA] = g_idata[cai + 2 * blockDim.x*blockIdx.x];// modification: data from the correct block
		temp[cbi + bankOffsetB] = g_idata[cbi + 2 * blockDim.x*blockIdx.x];
		for (int d = n >> 1; d > 0; d >>= 1) { // build sum in place up the tree  
			__syncthreads();
			if (thid < d) {
				int ai = offset*(2 * thid + 1) - 1;
				int bi = offset*(2 * thid + 2) - 1;
				ai += CONFLICT_FREE_OFFSET(ai);
				bi += CONFLICT_FREE_OFFSET(bi);
				temp[bi] += temp[ai];
			}
			offset *= 2;
		}
		if (thid == 0) {
			// store sum of block values
			g_udata[blockIdx.x] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
			temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
		}

		for (int d = 1; d < n; d *= 2) { // traverse down tree & build scan  
			offset >>= 1;
			__syncthreads();
			if (thid < d) {
				int ai = offset*(2 * thid + 1) - 1;
				int bi = offset*(2 * thid + 2) - 1;
				ai += CONFLICT_FREE_OFFSET(ai);
				bi += CONFLICT_FREE_OFFSET(bi);
				int t = temp[ai];
				temp[ai] = temp[bi];
				temp[bi] += t;
			}
		}
		__syncthreads();

		g_odata[cai + 2 * blockDim.x*blockIdx.x] = temp[cai + bankOffsetA];// modification: data to the correct block
		g_odata[cbi + 2 * blockDim.x*blockIdx.x] = temp[cbi + bankOffsetB];
	}
}

__global__ void add(unsigned int *out, unsigned int *in, int n) {
	int thid = threadIdx.x + blockDim.x* blockIdx.x;
	if (thid < n)
		out[thid] += in[blockIdx.x / 2];
}

static bool initCUDA(dev_prescan_t& dev) {
	CHECK_ERROR(hipMalloc((void**)&(dev.Idxs), dev.N*sizeof(unsigned int)));
	CHECK_ERROR(hipMemset(dev.Idxs, 0, dev.N*sizeof(unsigned int)));

	dev.blockCounts = (unsigned int*)malloc(dev.levels*sizeof(unsigned int));

	dev.count = getLengthAsPowersTwoSum(dev.blocks);
	dev.blockCounts[0] = dev.count;
	if ((dev.count - 1) / blockElements >= 1) {// how many times must be prescan run
		int dx = dev.count;
		int i = 1;
		while (dx > 1) {
			dx = dx / blockElements + (dx % blockElements > 0);
			dx = getLengthAsPowersTwoSum(dx);
			dev.blockCounts[i++] = dx;
			dev.count += dx;
		}
	}
	else {
		dev.count++;
		dev.blockCounts[1] = 1;
	}
#if DEBUG
	for (int i = 0; i < dev.levels; i++) {
		printf("%d ", dev.blockCounts[i]);
	}printf("\n");
	getchar();

	idxs = (unsigned int*)malloc(dev.N*sizeof(unsigned int));
	sums = (unsigned int*)malloc(dev.count*sizeof(unsigned int));
#endif
	CHECK_ERROR(hipMalloc((void**)&(dev.Sums), dev.count*sizeof(unsigned int)));
	CHECK_ERROR(hipMemset(dev.Sums, 0, dev.count*sizeof(unsigned int)));
	return true;
}

static bool callPrescan(dev_prescan_t& dev) {
	prescan<<<dev.blocks, THREADS_PER_BLOCK, 2 * THREADS_PER_BLOCK*sizeof(unsigned int)>>>(dev.Idxs, dev.Idxs, dev.Sums, dev.N);

#if DEBUG
	CHECK_ERROR(hipDeviceSynchronize());
	hipMemcpy(idxs, dev.Idxs, dev.N*sizeof(unsigned int), hipMemcpyDeviceToHost);
	for (int i = 0; i < dev.N; i++)	{
		printf("%d ", idxs[i]);
	}printf("\n");
	getchar();
#endif

	int dx, size, actBlocks;
	dev.count = dev.blockCounts[0];
	dev.startIdx = 0;
	for (int level = 1; level < dev.levels; level++) {// prescan of sums at each level
		dx = dev.blockCounts[level];
		size = dev.count - dev.startIdx;
		actBlocks = size / blockElements + (size % blockElements > 0);
#if DEBUG
		printf("%d %d %d %d %d\n", dx, dev.count, dev.startIdx, size, actBlocks);
#endif
		CHECK_ERROR(hipDeviceSynchronize());
		prescan<<<actBlocks, THREADS_PER_BLOCK, 2 * THREADS_PER_BLOCK*sizeof(unsigned int)>>>(
			dev.Sums + dev.startIdx, dev.Sums + dev.startIdx, dev.Sums + dev.count, size);
		CHECK_ERROR(hipGetLastError());
		dev.startIdx = dev.count;
		dev.count += dx;
	}

#if DEBUG
	hipMemcpy(sums, dev.Sums, dev.count*sizeof(unsigned int), hipMemcpyDeviceToHost);
	for (int i = 0; i < dev.count; i++)
	{
		printf("%d ", sums[i]);
	}printf("\n");
	getchar();
#endif
	return true;
}

static bool callAdd(int N, dev_prescan_t& dev) {
	int dx, level = dev.levels;
	if (level > 1) {
		dev.count -= dev.blockCounts[--level];
		dev.startIdx -= dev.blockCounts[level - 1];
	}
	while (level > 1) {
		dx = dev.blockCounts[--level];
		dev.count -= dx;
		dev.startIdx -= dev.blockCounts[level - 1];
#if DEBUG
		printf("%d %d %d\n", dx, count, startIdx);
#endif
		CHECK_ERROR(hipDeviceSynchronize());
		add<<<2 * dx, THREADS_PER_BLOCK>>>(dev.Sums + dev.startIdx, dev.Sums + dev.count, dev.count - dev.startIdx);
		CHECK_ERROR(hipGetLastError());
	}
	if (dev.blocks > 1) {// if all data are in one block, indexes are correct. otherwise add prescan sums
		CHECK_ERROR(hipDeviceSynchronize());
		add<<<2 * dev.blocks, THREADS_PER_BLOCK>>>(dev.Idxs, dev.Sums, N);
		CHECK_ERROR(hipGetLastError());
	}

#if DEBUG
	hipMemcpy(idxs, dev.Idxs, dev.N*sizeof(unsigned int), hipMemcpyDeviceToHost);
	for (int i = 0; i < dev.N; i++) {
		printf("%d ", idxs[i]);
	}printf("\n");
	getchar();
	hipMemcpy(sums, dev.Sums, dev.count*sizeof(unsigned int), hipMemcpyDeviceToHost);
	for (int i = 0; i < dev.count; i++) {
		printf("%d ", sums[i]);
	}printf("\n");
	getchar();
#endif
	return true;
}

static void freeCuda(dev_prescan_t& dev, bool onError) {

#define CUDA_FREE(ptr) if (ptr) {hipFree(ptr); ptr = nullptr;}
#define MEM_FREE(ptr) if (ptr) {free(ptr); ptr = nullptr;}

	if (onError) CUDA_FREE(dev.Idxs);// cannot be freed because it is returned
	CUDA_FREE(dev.Sums);

	if (dev.levels > 1) MEM_FREE(dev.blockCounts);

#if DEBUG
	MEM_FREE(sums);
	MEM_FREE(idxs);
#endif
}

unsigned int * prescan(int N, unsigned int * devArrIn) {
	dev_prescan_t dev;
	dev.blocks = (N % blockElements == 0) ? N / blockElements : N / blockElements + 1;
	dev.levels = (int)ceil(log((double)N) / log((double)blockElements));
	dev.N = getLengthAsPowersTwoSum(N);

	if (!initCUDA(dev)) return nullptr;
	hipMemcpy(dev.Idxs, devArrIn, N*sizeof(unsigned int), hipMemcpyDeviceToDevice);

	if (!callPrescan(dev)) return nullptr;
	if (!callAdd(N, dev)) return nullptr;
	if (IS_ERROR(hipDeviceSynchronize())) return nullptr;
	freeCuda(dev, false);

	return dev.Idxs;
}

